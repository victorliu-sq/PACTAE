#include <hip/hip_runtime.h>
#include <memory>
#include <smp/gs.cuh>
#include <smp/smp_engine_hybrid.cuh>
#include <glog/logging.h>
#include "utils/generate_worklods.h"
#include "smp/smp_engine_gs_2.h"
#include <set>

#include "smp_engine_test_utils.h"

int main(int argc, char *argv[]) {
  INIT_GLOG_STR("smp_engine_gs_par_test");


  SmpWorkloadConfig config;
  SetupSmpWorkloadConfig(config);

  TestSmpEngine<bamboosmp::SmpEngineGs2>(config);

  SHUTDOWN_GLOG();
}
